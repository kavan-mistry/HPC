

#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void cuda_hello_1_1() {
    printf("Hello World from GPU with grid dimension (1, 1) and block dimension (1, 1)!\n");
}

int main() {
    cuda_hello_1_1<<<1,1>>>();
    hipDeviceSynchronize(); // Make sure all GPU work is done before exiting
    return 0;
}
