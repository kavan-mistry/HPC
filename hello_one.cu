

#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void cuda_hello_one() {
    printf("Hello World from GPU !\n");
}

int main() {
    cuda_hello_one<<<1,4>>>();
    hipDeviceSynchronize(); // Make sure all GPU work is done before exiting
    return 0;
}
